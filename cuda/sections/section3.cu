#include "hip/hip_runtime.h"
#define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__ )

#include <iostream>
#include "../ray.cuh"
#include "../sphere.cuh"
#include <stdexcept>
#include <limits>

const double infinity = std::numeric_limits<double>::infinity();

// prints out any cuda errors that occur
void check_cuda(hipError_t result, const char *const func, const char *const file, const int line) {
  if (result) {
    std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " <<
              file << ":" << line << " '" << func << "'\n" <<
              hipGetErrorString(result)
              << "\n";
    // Make sure we call CUDA Device Reset before exiting
    hipDeviceReset();
    exit(99);
  }
}

__device__ vec3 ray_color(const ray& r) {
  sphere sp = sphere(point3(0,0,-1), 0.5);
  hit_record rec;
  if(sp.hit(r, 0, infinity, rec)) {
    return color(1,0,0);
  }
  vec3 unit_direction = unit_vector(r.direction());
  float t = 0.5f*(unit_direction.y() + 1.0f);
  return (1.0f-t)*vec3(1.0, 1.0, 1.0) + t*vec3(0.5, 0.7, 1.0);
}

__global__ void render(vec3 *fb, int max_x, int max_y, vec3 lower_left_corner, vec3 horizontal, vec3 vertical, vec3 origin) {
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  int j = threadIdx.y + blockIdx.y * blockDim.y;
  if((i >= max_x) || (j >= max_y)) return;
  int pixel_index = j*max_x + i;
  float u = float(i) / float(max_x);
  float v = float(j) / float(max_y);
  ray r(origin, lower_left_corner + u*horizontal + v*vertical);
  fb[pixel_index] = ray_color(r);
}

int main() {
  // Image
  const auto aspect_ratio = 16.0 / 9.0;
  const int nx = 400;
  const int ny = static_cast<int>(nx / aspect_ratio);

  // Camera
  auto viewport_height = 2.0;
  auto viewport_width = aspect_ratio * viewport_height;
  auto focal_length = 1.0;

  auto origin = point3(0, 0, 0);
  auto horizontal = vec3(viewport_width, 0, 0);
  auto vertical = vec3(0, viewport_height, 0);
  auto lower_left_corner = origin - horizontal/2 - vertical/2 - vec3(0, 0, focal_length);

  int num_pixels = nx*ny;
  size_t fb_size = num_pixels*sizeof(vec3);

  // allocate FB
  vec3 *fb;
  checkCudaErrors(hipMallocManaged((void **)&fb, fb_size));

  // one block is 64 threads
  int tx=8;
  int ty=8;

  dim3 blocks(nx/tx + 1, ny/ty + 1);
  // total threads are tx*ty
  dim3 threads(tx,ty);
  render<<<blocks, threads>>>(fb, nx, ny, lower_left_corner, horizontal, vertical, origin);
  checkCudaErrors(hipGetLastError());
  checkCudaErrors(hipDeviceSynchronize());

  // Output FB as Image
  std::cout << "P3\n" << nx << " " << ny << "\n255\n";
  for (int j = ny-1; j >= 0; j--) {
    for (int i = 0; i < nx; i++) {
      size_t pixel_index = j*nx + i;
      float r = fb[pixel_index].x();
      float g = fb[pixel_index].y();
      float b = fb[pixel_index].z();
      int ir = int(255.99*r);
      int ig = int(255.99*g);
      int ib = int(255.99*b);
      std::cout << ir << " " << ig << " " << ib << "\n";
    }
  }
  checkCudaErrors(hipFree(fb));

  return 0;
}
