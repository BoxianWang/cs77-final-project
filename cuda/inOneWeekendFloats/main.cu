#include "hip/hip_runtime.h"
#define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__ )

#include <iostream>
#include "ray.cuh"
#include "objects/sphere.cuh"
#include "objects/hittable.cuh"
#include "objects/hittable_list.cuh"
#include <stdexcept>
#include <limits>
#include <hiprand/hiprand_kernel.h>
#include "camera.cuh"
#include "randoms.cuh"
#include "materials/material.cuh"
#include "materials/lambertian.cuh"
#include "materials/metal.cuh"
#include "materials/dielectric.cuh"

const float infinity = std::numeric_limits<float>::infinity();

// prints out any cuda errors that occur
void check_cuda(hipError_t result, const char *const func, const char *const file, const int line) {
  if (result) {
    std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " <<
              file << ":" << line << " '" << func << "'\n" <<
              hipGetErrorString(result)
              << "\n";
    // Make sure we call CUDA Device Reset before exiting
    hipDeviceReset();
    exit(99);
  }
}

// creates the world -- both the materials and the spheres
__global__ void create_world(hittable **d_list, hittable **d_world, hiprandState *rand_state) {
  if (threadIdx.x == 0 && blockIdx.x == 0) {
    hiprandState* local_rand_state = rand_state;

    auto material_ground = new lambertian(vec3(0.5, 0.5, 0.5));
    auto material_1   = new dielectric(1.5);
    auto material_2 = new lambertian(vec3(0.4, 0.2, 0.1));
    auto material_3  = new metal(vec3(0.7, 0.6, 0.5), 0.);

    int sphereNum = 0;
    d_list[sphereNum++] = new sphere(vec3(0,-1000,0), 1000, material_ground);
    d_list[sphereNum++] = new sphere(vec3(0, 1, 0), 1, material_1);
    d_list[sphereNum++] = new sphere(vec3(-4, 1, 0), 1, material_2);
    d_list[sphereNum++] = new sphere(vec3(4, 1, 0), 1, material_3);

    for (int a = -11; a < 11; a++) {
      for (int b = -11; b < 11; b++) {
        float choose_mat = random_float(local_rand_state);
        point3 center(float(a) + 0.f*random_float(local_rand_state), 0.2f, float(b) + 0.9f*random_float(local_rand_state));

        if ((center - point3(4, 0.2, 0)).length() > 0.9) {
          material* sphere_material;

          if (choose_mat < 0.8) {
            // diffuse
            auto albedo = vec3_random(local_rand_state) * vec3_random(local_rand_state);
            sphere_material = new lambertian(albedo);
            d_list[sphereNum++] = new sphere(center, 0.2, sphere_material);
          } else if (choose_mat < 0.95) {
            // metal
            auto albedo = vec3_random(local_rand_state, 0.5, 1);
            auto fuzz = random_float(local_rand_state, 0, 0.5);
            sphere_material = new metal(albedo, fuzz);
            d_list[sphereNum++] = new sphere(center, 0.2, sphere_material);
          } else {
            // glass
            sphere_material = new dielectric(1.5);
            d_list[sphereNum++] = new sphere(center, 0.2, sphere_material);
          }
        }
      }
    }

    *d_world = new hittable_list(d_list, sphereNum);
  }
}

// cleans up the world
__global__ void free_world(hittable **d_list, hittable **d_world) {
  int objectNumber = (*d_world)->getObjectNumber();
  for (int i = 0; i < objectNumber; i++) {
    delete *(d_list+i);
  }
  delete *d_world;
}

// creates the camera
__global__ void create_camera(camera **d_cam) {
  if (threadIdx.x == 0 && blockIdx.x == 0) {
    auto origin = point3(13, 2, 3);
    auto lookAt = point3(0, 0, 0);
    auto vup = point3(0, 1.0, 0);
    float vFov = 20;
    float aspectRatio = 3./2.;
    float aperture = .1;
    float dist_to_focus = 10;
    *d_cam = new camera(origin, lookAt, vup, vFov, aspectRatio, aperture, dist_to_focus);
  }
}

// deletes the camera
__global__ void free_camera(camera **d_cam) {
  delete *(d_cam);
}

// initializes the rand state for memory
__global__ void render_init(int max_x, int max_y, hiprandState *rand_state) {
  unsigned int i = threadIdx.x + blockIdx.x * blockDim.x;
  unsigned int j = threadIdx.y + blockIdx.y * blockDim.y;
  if((i >= max_x) || (j >= max_y)) return;
  int pixel_index = int(j)*max_x + int(i);
  //Each thread gets same seed, a different sequence number, no offset
  hiprand_init(1984, pixel_index, 0, &rand_state[pixel_index]);
}

// colors the ray
__device__ vec3 ray_color(const ray& r, hittable **world, hiprandState *local_rand_state) {
  ray cur_ray = r;
  vec3 cur_attenuation = vec3(1,1,1);
  for(int i = 0; i < 50; i++) {
    hit_record rec;
    if ((*world)->hit(cur_ray, 0.001f, infinity, rec)) {
      ray scattered;
      color new_attenuation;
      if (rec.mat_ptr->scatter(cur_ray, rec, new_attenuation, scattered, local_rand_state)) {
        cur_attenuation = new_attenuation * cur_attenuation;
        cur_ray = scattered;
      }
    }
    else {
      // light stops bouncing
      vec3 unit_direction = unit_vector(cur_ray.direction());
      float t = 0.5f*(unit_direction.y() + 1.0f);
      vec3 c = (1.0f-t)*vec3(1.0, 1.0, 1.0) + t*vec3(0.5, 0.7, 1.0);
      return cur_attenuation * c;
    }
  }
  return vec3(0.0,0.0,0.0); // exceeded recursion
}

// actually renders a pixel
__global__ void render(
    vec3 *fb, int max_x, int max_y, hittable** world, hiprandState *rand_state,
    camera** cam,
    int number_samples
) {
  unsigned int i = threadIdx.x + blockIdx.x * blockDim.x;
  unsigned int j = threadIdx.y + blockIdx.y * blockDim.y;
  if((i >= max_x) || (j >= max_y)) return;
  int pixel_index = int(j)*max_x + int(i);
  hiprandState local_rand_state = rand_state[pixel_index];

  fb[pixel_index] = vec3(0,0,0);

  // this loop handles super sampling for antialiasing
  for (int it = 0; it < number_samples; it++) {
    float u = (float(i) + hiprand_uniform(&local_rand_state)) / float(max_x);
    float v = (float(j) + hiprand_uniform(&local_rand_state)) / float(max_y);
    ray r = (*cam)->get_ray(&local_rand_state, u, v);
    fb[pixel_index] += ray_color(r, world, &local_rand_state) / float(number_samples);
  }

  // gamma correct
  fb[pixel_index] = vec3(sqrt(fb[pixel_index].x()),
                         sqrt(fb[pixel_index].y()),
                         sqrt(fb[pixel_index].z())
                         );
}

int main() {
  // Image
  const auto aspect_ratio = 3.0 / 2.0;
  const int nx = 1200;
  const int ny = static_cast<int>(nx / aspect_ratio);
  std::cerr << nx << " by " << ny << " image\n";

  // Camera

  int num_pixels = nx*ny;
  size_t fb_size = num_pixels*sizeof(vec3);

  // allocate FB
  vec3 *fb;
  checkCudaErrors(hipMallocManaged((void **)&fb, fb_size));

  // allocate cuda rand state
  hiprandState *d_rand_state;
  checkCudaErrors(hipMalloc((void **)&d_rand_state, num_pixels*sizeof(hiprandState)));

  // one block is 64 threads
  int tx=8;
  int ty=8;

  // set up the random state
  dim3 blocks(nx/tx + 1, ny/ty + 1);
  // total threads are tx*ty
  dim3 threads(tx,ty);
  // initialize the rand state
  render_init<<<blocks, threads>>>(nx, ny, d_rand_state);

  // generate the world
  hittable **d_list;
  checkCudaErrors(hipMalloc((void **)&d_list, 488*sizeof(hittable *)));
  hittable **d_world;
  checkCudaErrors(hipMalloc((void **)&d_world, sizeof(hittable *)));
  create_world<<<1,1>>>(d_list, d_world, d_rand_state);
  checkCudaErrors(hipGetLastError());
  checkCudaErrors(hipDeviceSynchronize());

  // generate the camera
  camera **d_cam;
  checkCudaErrors(hipMalloc((void **)&d_cam, sizeof(camera*)));
  create_camera<<<1,1>>>(d_cam);
  checkCudaErrors(hipGetLastError());
  checkCudaErrors(hipDeviceSynchronize());

  std::cerr << "Rendering...";

  clock_t start, stop;
  start = clock();

  std::cerr << "\nInit fin...\n";
  checkCudaErrors(hipGetLastError());
  checkCudaErrors(hipDeviceSynchronize());
  render<<<blocks, threads>>>(fb, nx, ny,
                              d_world,
                              d_rand_state,
                              d_cam,
                              500       // number_samples
                              );
  checkCudaErrors(hipGetLastError());
  checkCudaErrors(hipDeviceSynchronize());

  stop = clock();
  float timer_seconds = ((float)(stop - start)) / CLOCKS_PER_SEC;
  std::cerr << "took " << timer_seconds << " seconds.\n";

  // Output FB as Image
  std::cout << "P3\n" << nx << " " << ny << "\n255\n";
  for (int j = ny-1; j >= 0; j--) {
    for (int i = 0; i < nx; i++) {
      size_t pixel_index = j*nx + i;
      float r = fb[pixel_index].x();
      float g = fb[pixel_index].y();
      float b = fb[pixel_index].z();
      int ir = int(255.99*r);
      int ig = int(255.99*g);
      int ib = int(255.99*b);
      std::cout << ir << " " << ig << " " << ib << "\n";
    }
  }
  // clean up
  free_world<<<1,1>>>(d_list,d_world);
  free_camera<<<1,1>>>(d_cam);
  checkCudaErrors(hipGetLastError());
  checkCudaErrors(hipFree(d_list));
  checkCudaErrors(hipFree(d_world));
  checkCudaErrors(hipFree(d_cam));
  checkCudaErrors(hipFree(fb));

  return 0;
}
