#include "hip/hip_runtime.h"
#define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__ )

#include <iostream>
#include "ray.cuh"
#include "objects/sphere.cuh"
#include "objects/moving_sphere.cuh"
#include "objects/hittable.cuh"
#include "objects/box.cuh"
#include "objects/bumpy_sphere.cuh"
#include "objects/hittable_list.cuh"
#include "objects/aarect.cuh"
#include <stdexcept>
#include <limits>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime.h>
#include "camera.cuh"
#include "randoms.cuh"
#include "rtw_stb_image.cuh"
#include "texture/texture.cuh"
#include "texture/image.cuh"
#include "texture/checker.cuh"
#include "texture/noise.cuh"
#include "materials/material.cuh"
#include "materials/lambertian.cuh"
#include "materials/metal.cuh"
#include "materials/dielectric.cuh"
#include "materials/diffuse.cuh"
#include "objects/constant_medium.cuh"




// prints out any cuda errors that occur
void check_cuda(hipError_t result, const char *const func, const char *const file, const int line) {
  if (result) {
    std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " <<
              file << ":" << line << " '" << func << "'\n" <<
              hipGetErrorString(result)
              << "\n";
    // Make sure we call CUDA Device Reset before exiting
    hipDeviceReset();
    exit(99);
  }
}

__host__ void prepare_texture(int *height, int *width, char * filename) {
    unsigned char *data;

    int components_per_pixel = 3;
    data = stbi_load(
            filename, width, height, &components_per_pixel, components_per_pixel);
    int length = *width * *height* 3;

    // build 1D texture 
    size_t offset = 0;
    tex.addressMode[0] = hipAddressModeBorder;
    tex.addressMode[1] = hipAddressModeBorder;
    tex.filterMode = hipFilterModePoint;
    tex.normalized = false;

    unsigned int* ddata;

    checkCudaErrors(hipMalloc((void**)&ddata, sizeof(unsigned char)*length));
    hipMemcpy(ddata, data, sizeof(unsigned char)*length, hipMemcpyHostToDevice);

    hipBindTexture(&offset, tex, ddata, sizeof(unsigned char)*length);
    checkCudaErrors(hipGetLastError());
}

__global__ void two_perlin_spheres(hittable **d_list, hittable **d_world, hiprandState *rand_state) {
  if (threadIdx.x == 0 && blockIdx.x == 0) {
    auto pertext = new noise_texture(4, rand_state);
    auto mat = new lambertian(pertext);

    int sphereNum = 0;
    d_list[sphereNum++] = new sphere(point3(0,-1000,0), 1000, mat);
    d_list[sphereNum++] = new sphere(point3(0, 2, 0), 2, mat);

    *d_world = new hittable_list(d_list, sphereNum, rand_state);
  }
}

__global__ void two_perlin_spheres_2(hittable **d_list, hittable **d_world, hiprandState *rand_state) {
  if (threadIdx.x == 0 && blockIdx.x == 0) {
    auto pertext = new noise_texture(10, rand_state, 1);
    auto mat = new lambertian(pertext);

    int sphereNum = 0;
    d_list[sphereNum++] = new sphere(point3(0,-1000,0), 1000, mat);
    d_list[sphereNum++] = new sphere(point3(0, 2, 0), 2, mat);

    *d_world = new hittable_list(d_list, sphereNum, rand_state);
  }
  // if (threadIdx.x == 0 && blockIdx.x == 0) {
  //   auto pertext = new noise_texture(20, rand_state, 1);
  //   auto mat = new lambertian(pertext);

  //   int sphereNum = 0;

  //   d_list[sphereNum++] = new sphere(point3(0, 9, 0), 2, mat);

  //   *d_world = new hittable_list(d_list, sphereNum, rand_state);
  // }
}


__global__ void two_bumpy_spheres(hittable **d_list, hittable **d_world, hiprandState *rand_state) {
  if (threadIdx.x == 0 && blockIdx.x == 0) {
    auto pertext = new solid_color(color(0.3,0.3,0.3));
    auto mat = new lambertian(pertext);
    auto mat2 = new lambertian(color(0.8,0.8,0.8));

    int sphereNum = 0;
    d_list[sphereNum++] = new sphere(point3(0,-1000,0), 1000, mat2);
    d_list[sphereNum++] = new bumpy_sphere(point3(0, 2, 0), 2, mat, rand_state);

    *d_world = new hittable_list(d_list, sphereNum, rand_state);
  }
}

__global__ void earth(hittable **d_list, hittable **d_world, hiprandState *rand_state, int height, int width) {
  if (threadIdx.x == 0 && blockIdx.x == 0) {
    auto pertext = new noise_texture(4, rand_state);
    auto mat = new lambertian(pertext);

    int sphereNum = 0;
    d_list[sphereNum++] = new sphere(point3(0,-1000,0), 1000, mat);


    // build sphere
    auto imtext = new image_texture(width, height);
    auto mat2 = new lambertian(imtext);

    d_list[sphereNum++] = new sphere(point3(0,2,0), 2, mat2);

    *d_world = new hittable_list(d_list, sphereNum, rand_state);
  }
}

__global__ void cornell_box(hittable **d_list, hittable **d_world, hiprandState *rand_state) {
  if (threadIdx.x == 0 && blockIdx.x == 0) {


    // build sphere

    int sphereNum = 0;
    auto red   = new  lambertian(color(.65, .05, .05));
    auto white = new  lambertian(color(.73, .73, .73));
    auto green = new  lambertian(color(.12, .45, .15));
    auto light = new diffuse_light(color(15, 15, 15));

    d_list[sphereNum++] = new yz_rect(0, 555, 0, 555, 555, green);
    d_list[sphereNum++] = new yz_rect(0, 555, 0, 555, 0, red);
    d_list[sphereNum++] = new xz_rect(213, 343, 227, 332, 554, light);
    d_list[sphereNum++] = new xz_rect(0, 555, 0, 555, 0, white);
    d_list[sphereNum++] = new xz_rect(0, 555, 0, 555, 555, white);
    d_list[sphereNum++] = new xy_rect(0, 555, 0, 555, 555, white);

    hittable **side1 = (hittable **)malloc(sizeof(hittable*)*6);
    hittable **side2 = (hittable **)malloc(sizeof(hittable*)*6);
    hittable *box1 = new box(point3(0, 0, 0), point3(165, 330, 165), white, side1, rand_state);
    hittable *box2 = new box(point3(0,0,0), point3(165,165,165), white, side2, rand_state);
    d_list[sphereNum++] = new translate(new rotate_y(box1, 15), vec3(265,0,295));
    d_list[sphereNum++] = new translate(new rotate_y(box2, -18), vec3(130,0,65));

    *d_world = new hittable_list(d_list, sphereNum, rand_state);
  }
}

__global__ void cornell_smoke(hittable **d_list, hittable **d_world, hiprandState *rand_state) {
  if (threadIdx.x == 0 && blockIdx.x == 0) {


    // build sphere

    int sphereNum = 0;
    auto red   = new  lambertian(color(.65, .05, .05));
    auto white = new  lambertian(color(.73, .73, .73));
    auto green = new  lambertian(color(.12, .45, .15));
    auto light = new diffuse_light(color(7, 7, 7));

    d_list[sphereNum++] = new yz_rect(0, 555, 0, 555, 555, green);
    d_list[sphereNum++] = new yz_rect(0, 555, 0, 555, 0, red);
    d_list[sphereNum++] = new xz_rect(113, 443, 127, 432, 554, light);
    d_list[sphereNum++] = new xz_rect(0, 555, 0, 555, 0, white);
    d_list[sphereNum++] = new xz_rect(0, 555, 0, 555, 555, white);
    d_list[sphereNum++] = new xy_rect(0, 555, 0, 555, 555, white);

    hittable **side1 = (hittable **)malloc(sizeof(hittable*)*6);
    hittable **side2 = (hittable **)malloc(sizeof(hittable*)*6);
    hittable *box1 = new box(point3(0, 0, 0), point3(165, 330, 165), white, side1, rand_state);
    hittable *box2 = new box(point3(0,0,0), point3(165,165,165), white, side2, rand_state);
    box1 = new translate(new rotate_y(box1, 15), vec3(265,0,295));
    box2 = new translate(new rotate_y(box2, -18), vec3(130,0,65));
    d_list[sphereNum++] = new constant_medium(rand_state, box1, 0.01, color(0,0,0));
    d_list[sphereNum++] = new constant_medium(rand_state, box2, 0.01, color(1,1,1));

    *d_world = new hittable_list(d_list, sphereNum, rand_state);
  }
}

__global__ void simple_light(hittable **d_list, hittable **d_world, hiprandState *rand_state) {
  if (threadIdx.x == 0 && blockIdx.x == 0) {


    // build sphere
    auto pertext = new noise_texture(4, rand_state);
    auto mat = new lambertian(pertext);

    int sphereNum = 0;
    d_list[sphereNum++] = new sphere(point3(0,-1000,0), 1000, mat);
    d_list[sphereNum++] = new sphere(point3(0, 2, 0), 2, mat);

    auto diff = new diffuse_light(color(4,4,4));
    // auto diff  = new metal(vec3(0.7, 0.6, 0.5), 0.);
    d_list[sphereNum++] = new xy_rect(3, 5, 1, 3, -2, diff);

    d_list[sphereNum++] = new sphere(point3(0, 7, 0), 2, diff);

    *d_world = new hittable_list(d_list, sphereNum, rand_state);
  }
}

// creates the world -- both the materials and the spheres
__global__ void random_world(hittable **d_list, hittable **d_world, hiprandState *rand_state) {
  if (threadIdx.x == 0 && blockIdx.x == 0) {
    hiprandState* local_rand_state = rand_state;
    auto checker = new checker_texture(color(0.2, 0.3, 0.1), color(0.9, 0.9, 0.9));
    auto material_ground = new lambertian(checker);

    auto material_1   = new dielectric(1.5);
    auto material_2 = new lambertian(vec3(0.4, 0.2, 0.1));
    auto material_3  = new metal(vec3(0.7, 0.6, 0.5), 0.);

    int sphereNum = 0;
    d_list[sphereNum++] = new sphere(vec3(0,-1000,0), 1000, material_ground);
    d_list[sphereNum++] = new sphere(vec3(0, 1, 0), 1, material_1);
    d_list[sphereNum++] = new sphere(vec3(-4, 1, 0), 1, material_2);
    d_list[sphereNum++] = new sphere(vec3(4, 1, 0), 1, material_3);

    for (int a = -11; a < 11; a++) {
      for (int b = -11; b < 11; b++) {
        float choose_mat = random_float(local_rand_state);
        point3 center(float(a) + 0.f*random_float(local_rand_state), 0.2f, float(b) + 0.9f*random_float(local_rand_state));

        if ((center - point3(4, 0.2, 0)).length() > 0.9) {
          material* sphere_material;

          if (choose_mat < 0.8) {
            // diffuse
            auto albedo = vec3_random(local_rand_state) * vec3_random(local_rand_state);
            sphere_material = new lambertian(albedo);
            vec3 center2 = center + vec3(0.f, random_float(local_rand_state, 0.f, .5f), 0.f);
            d_list[sphereNum++] = new moving_sphere(center, center2, 0., 1., 0.2, sphere_material);
            //d_list[sphereNum++] = new sphere(center, 0.2, sphere_material);
          } else if (choose_mat < 0.95) {
            // metal
            auto albedo = vec3_random(local_rand_state, 0.5, 1);
            auto fuzz = random_float(local_rand_state, 0, 0.5);
            sphere_material = new metal(albedo, fuzz);
            d_list[sphereNum++] = new sphere(center, 0.2, sphere_material);
          } else {
            // glass
            sphere_material = new dielectric(1.5);
            d_list[sphereNum++] = new sphere(center, 0.2, sphere_material);
          }
        }
      }
    }

    *d_world = new hittable_list(d_list, sphereNum, local_rand_state);
  }
}

__global__ void final_scene(hittable **d_list, hittable **d_world, hiprandState *rand_state, int height, int width) {
  if (threadIdx.x == 0 && blockIdx.x == 0) {
    int sphereNum = 0;
    auto ground = new lambertian(color(0.48, 0.83, 0.53));

    const int boxes_per_side = 20;
    for (int i = 0; i < boxes_per_side; i++) {
        for (int j = 0; j < boxes_per_side; j++) {
            auto w = 100.0;
            auto x0 = -1000.0 + i*w;
            auto z0 = -1000.0 + j*w;
            auto y0 = 0.0;
            auto x1 = x0 + w;
            auto y1 = random_float(rand_state, 1,101);
            auto z1 = z0 + w;
            hittable **side = (hittable **)malloc(sizeof(hittable*)*6);
            d_list[sphereNum++] = (new box(point3(x0,y0,z0), point3(x1,y1,z1), ground,  side, rand_state));
        }
    }



    auto light = new diffuse_light(color(7, 7, 7));
    d_list[sphereNum++] =(new xz_rect(123, 423, 147, 412, 554, light));

    auto center1 = point3(400, 400, 200);
    auto center2 = center1 + vec3(30,0,0);
    auto moving_sphere_material = new lambertian(color(0.7, 0.3, 0.1));
    d_list[sphereNum++] =(new moving_sphere(center1, center2, 0, 1, 50, moving_sphere_material));

    d_list[sphereNum++] =(new sphere(point3(260, 150, 45), 50, new dielectric(1.5)));
    d_list[sphereNum++] =(new sphere(
        point3(0, 150, 145), 50, new metal(color(0.8, 0.8, 0.9), 1.0)
    ));

    auto boundary = new sphere(point3(360,150,145), 70, new dielectric(1.5));
    d_list[sphereNum++] =(boundary);
    d_list[sphereNum++] =(new constant_medium(rand_state, boundary, 0.2, color(0.2, 0.4, 0.9)));
    boundary = new sphere(point3(0, 0, 0), 5000, new dielectric(1.5));
    d_list[sphereNum++] =(new constant_medium(rand_state, boundary, .0001, color(1,1,1)));


    auto emat = new lambertian( new image_texture(width, height));
    d_list[sphereNum++] =(new sphere(point3(400,200,400), 100, emat));
    auto pertext = new noise_texture(0.1, rand_state);
    d_list[sphereNum++] =(new sphere(point3(220,280,300), 80, new lambertian(pertext)));

    auto white = new lambertian(color(.73, .73, .73));
    int ns = 1000;
    for (int j = 0; j < ns; j++) {
        d_list[sphereNum++] =(new sphere(vec3_random(rand_state, 0,165), 10, white));
    }

    *d_world = new hittable_list(d_list, sphereNum, rand_state);
  }
}

// cleans up the world
__global__ void free_world(hittable **d_list, hittable **d_world) {
  int objectNumber = (*d_world)->getObjectNumber();
  for (int i = 0; i < objectNumber; i++) {
    delete *(d_list+i);
  }
  delete *d_world;
}

// creates the camera
__global__ void create_camera(camera **d_cam) {
  if (threadIdx.x == 0 && blockIdx.x == 0) {
    vec3 origin, lookAt, vup;
    float vFov, aspectRatio, aperture, dist_to_focus;
    switch (0)
    {
    case 0:
    // perlin
      origin = point3(13, 2, 3);
      lookAt = point3(0, 0, 0);
      vup = point3(0, 1.0, 0);
      vFov = 20;
      aspectRatio = 3./2.;
      aperture = .0;
      dist_to_focus = 10;
      break;
    
    default:
    case 1:
    // random
      origin = point3(13, 2, 3);
      lookAt = point3(0, 0, 0);
      vup = point3(0, 1.0, 0);
      vFov = 20;
      aspectRatio = 3./2.;
      aperture = .1;
      dist_to_focus = 10;
      
      break;
    case 3:
      origin = point3(26,3,6);
      lookAt = point3(0,2,0);
      vup = point3(0, 1.0, 0);
      vFov = 20;
      aspectRatio = 3./2.;
      aperture = .0;
      dist_to_focus = 10;
      break;
    case 4:
      origin = point3(278, 278, -800);
      lookAt = point3(278, 278, 0);
      vup = point3(0, 1.0, 0);
      vFov = 40;
      aspectRatio = 1.0;
      dist_to_focus = 10;
      break;
    case 5:
    // custom
      origin = point3(0, 0, 0);
      lookAt = point3(0, 1, 0);
      vup = point3(0, 0, 1);
      vFov = 40;
      aspectRatio = 3./2.;
      aperture = .0;
      dist_to_focus = 6;
      break;
    case 6:
    // fina scene
      origin = point3(478, 278, -600);
      lookAt = point3(278, 278, 0);
      vup = point3(0, 1.0, 0);
      vFov = 40.0;
      aperture = 0.0;
      dist_to_focus = 10.0;
      break;
    

    }
    *d_cam = new camera(origin, lookAt, vup, vFov, aspectRatio, aperture, dist_to_focus, 0., 1.);
  }
}


// deletes the camera
__global__ void free_camera(camera **d_cam) {
  delete *(d_cam);
}

// initializes the rand state for memory
__global__ void render_init(int max_x, int max_y, hiprandState *rand_state) {
  unsigned int i = threadIdx.x + blockIdx.x * blockDim.x;
  unsigned int j = threadIdx.y + blockIdx.y * blockDim.y;
  if((i >= max_x) || (j >= max_y)) return;
  int pixel_index = int(j)*max_x + int(i);
  //Each thread gets same seed, a different sequence number, no offset
  hiprand_init(clock64(), pixel_index, 0, &rand_state[pixel_index]);
}

// colors the ray
__device__ vec3 ray_color(const ray& r, const color& background, hittable **world, hiprandState *local_rand_state) {
  ray cur_ray = r;
  vec3 cur_prod = vec3(1,1,1);
  vec3 cur_sum = vec3(0,0,0);
  int max_depth = 100;
  for(int i = 0; i < max_depth; i++) {
    hit_record rec;
    if ((*world)->hit(cur_ray, 0.001f, infinity, rec)) {
      ray scattered;
      color new_attenuation;
      color emitted = rec.mat_ptr->emitted(rec.u, rec.v, rec.p);
      if (rec.mat_ptr->scatter(cur_ray, rec, new_attenuation, scattered, local_rand_state)) {
        cur_sum = cur_sum + emitted * cur_prod;
        cur_prod = cur_prod * new_attenuation;
        cur_ray = scattered;
      } else return cur_sum + cur_prod * emitted;
    }
    else {
      // light stops bouncing
      return cur_sum + cur_prod * background;
    }
  }
  return cur_sum; // exceeded recursion
}




// actually renders a pixel
__global__ void render(
    vec3 *fb, int max_x, int max_y, hittable** world, hiprandState *rand_state,
    camera** cam,
    int number_samples, color background
) {
  unsigned int i = threadIdx.x + blockIdx.x * blockDim.x;
  unsigned int j = threadIdx.y + blockIdx.y * blockDim.y;
  if((i >= max_x) || (j >= max_y)) return;
  int pixel_index = int(j)*max_x + int(i);
  hiprandState local_rand_state = rand_state[pixel_index];

  fb[pixel_index] = vec3(0,0,0);

  // this loop handles super sampling for antialiasing
  for (int it = 0; it < number_samples; it++) {
    float u = (float(i) + hiprand_uniform(&local_rand_state)) / float(max_x);
    float v = (float(j) + hiprand_uniform(&local_rand_state)) / float(max_y);
    ray r = (*cam)->get_ray(&local_rand_state, u, v);
    fb[pixel_index] += ray_color(r, background, world, &local_rand_state);
  }

  fb[pixel_index] /= float(number_samples);

  // gamma correct
  fb[pixel_index] = vec3(__saturatef(sqrt(fb[pixel_index].x())),
                         __saturatef(sqrt(fb[pixel_index].y())),
                         __saturatef(sqrt(fb[pixel_index].z()))
                         );
}

int main() {
  // Image
  float aspect_ratio;
  int nx;
  switch (0)
  {
  case 0:
    aspect_ratio = 3.0/2.0;
    nx = 1200;
    break;
  
  default:
  // cornell
  case 1:
    aspect_ratio = 1.0;
    nx = 600;
    break;
  }
  
  const int ny = static_cast<int>(nx / aspect_ratio);
  std::cerr << nx << " by " << ny << " image\n";

  size_t size = 0;
  cudaThreadSetLimit(hipLimitStackSize, 4096);
  cudaThreadGetLimit(&size, hipLimitStackSize);
  std::cerr << "STACK SIZE: " << size << std::endl;
  // Camera

  int num_pixels = nx*ny;
  size_t fb_size = num_pixels*sizeof(vec3);

  // allocate FB
  vec3 *fb;
  checkCudaErrors(hipMallocManaged((void **)&fb, fb_size));

  // allocate cuda rand state
  hiprandState *d_rand_state;
  checkCudaErrors(hipMalloc((void **)&d_rand_state, num_pixels*sizeof(hiprandState)));

  // one block is 64 threads
  int tx=8;
  int ty=8;

  // set up the random state
  dim3 blocks(nx/tx + 1, ny/ty + 1);
  // total threads are tx*ty
  dim3 threads(tx,ty);
  // initialize the rand state
  render_init<<<blocks, threads>>>(nx, ny, d_rand_state);

  // bg color
  color background(0, 0, 0);
  int num_samples = 100;
  int width, height;  //img texture

  hittable **d_list;
  hittable **d_world;
  camera **d_cam;

  checkCudaErrors(hipMalloc((void **)&d_list, 500*sizeof(hittable *)));
  checkCudaErrors(hipDeviceSynchronize());
  checkCudaErrors(hipMalloc((void **)&d_world, sizeof(hittable *)));
  checkCudaErrors(hipDeviceSynchronize());


  // choose a scene
  switch (7)
  {
  case 0:
    two_perlin_spheres<<<1,1>>>(d_list, d_world, d_rand_state);
    background = color(0.70, 0.80, 1.00);
    num_samples = 200;
    break;
  
  default:
  case 1:
      // generate the world
    random_world<<<1,1>>>(d_list, d_world, d_rand_state);
    background = color(0.70, 0.80, 1.00);

    break;
  case 2:
      // read file
    prepare_texture(&height, &width, "../earthmap.jpg");

    earth<<<1,1>>>(d_list, d_world, d_rand_state, height, width);
    background = color(0.70, 0.80, 1.00);

    break;
  case 3:
    simple_light<<<1,1>>>(d_list, d_world, d_rand_state);
    background = color(0,0,0);
    //  background = color(0.70, 0.80, 1.00);
    num_samples = 400;
    break;
  
  case 4:
    cornell_box<<<1,1>>>(d_list, d_world, d_rand_state);
    background = color(0,0,0);
    num_samples = 400;
    break;
  case 5:
    cornell_smoke<<<1,1>>>(d_list, d_world, d_rand_state);
    background = color(0,0,0);
    num_samples = 400;
    break;
  case 6:
    // circular noise
    two_perlin_spheres_2<<<1,1>>>(d_list, d_world, d_rand_state);
    background = color(0.70, 0.80, 1.00);
    break;
  case 7:
    // bump mapping
    two_bumpy_spheres<<<1,1>>>(d_list, d_world, d_rand_state);
    background = color(0.70, 0.80, 1.00);
    num_samples = 200;
    break;
  case 8:
    // final scene (work in progress)
    prepare_texture(&height, &width, "../earthmap.jpg");
    final_scene<<<1,1>>>(d_list, d_world, d_rand_state, height, width);
    background = color(0.70, 0.80, 1.00);
    num_samples = 400;
    break;
  }
checkCudaErrors(hipGetLastError());
checkCudaErrors(hipDeviceSynchronize());

      // generate the camera
    
  checkCudaErrors(hipMalloc((void **)&d_cam, sizeof(camera*)));
  create_camera<<<1,1>>>(d_cam);
  checkCudaErrors(hipGetLastError());
  checkCudaErrors(hipDeviceSynchronize());

  // reset the stack size
  cudaThreadSetLimit(hipLimitStackSize, 1024);

  std::cerr << "Rendering...";

  clock_t start, stop;
  start = clock();

  std::cerr << "\nInit fin...\n";
  checkCudaErrors(hipGetLastError());
  checkCudaErrors(hipDeviceSynchronize());
  render<<<blocks, threads>>>(fb, nx, ny,
                              d_world,
                              d_rand_state,
                              d_cam,
                              num_samples, background      // number_samples
                              );
  checkCudaErrors(hipGetLastError());
  checkCudaErrors(hipDeviceSynchronize());

  stop = clock();
  float timer_seconds = ((float)(stop - start)) / CLOCKS_PER_SEC;
  std::cerr << "took " << timer_seconds << " seconds.\n";

  // Output FB as Image
  std::cout << "P3\n" << nx << " " << ny << "\n255\n";
  for (int j = ny-1; j >= 0; j--) {
    for (int i = 0; i < nx; i++) {
      size_t pixel_index = j*nx + i;
      float r = fb[pixel_index].x();
      float g = fb[pixel_index].y();
      float b = fb[pixel_index].z();
      int ir = int(255.99*r);
      int ig = int(255.99*g);
      int ib = int(255.99*b);
      std::cout << ir << " " << ig << " " << ib << "\n";
    }
  }
  // clean up
  free_world<<<1,1>>>(d_list,d_world);
  free_camera<<<1,1>>>(d_cam);
  checkCudaErrors(hipGetLastError());
  checkCudaErrors(hipFree(d_list));
  checkCudaErrors(hipFree(d_world));
  checkCudaErrors(hipFree(d_cam));
  checkCudaErrors(hipFree(fb));

  return 0;
}
